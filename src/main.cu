/*
 * main.cu
 *
 *  Created on: Nov 13, 2015
 *      Author: reneoctavio
 */

#include "definitions.cuh"

int main(int argc, char* argv[]) {
	if (argc != 4) return 1;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	const int k_warp_size = prop.warpSize;

	const int k_avg_degree = std::atoi(argv[1]);
	const int k_avg_edge_length = std::atoi(argv[2]);

	const int k_delta = k_warp_size * k_avg_edge_length / k_avg_degree;

	std::cout << "Device: " << prop.name << std::endl;
	std::cout << "Warp Size: " << prop.warpSize << std::endl;
	std::cout << "Average degree: " << k_avg_degree << std::endl;
	std::cout << "Average edge length: " << k_avg_edge_length << std::endl;
	std::cout << "Calculated delta: " << k_delta << std::endl;

	DCsrMatrix d_graph;
	cusp::io::read_dimacs_file(d_graph, argv[3]);

	const int k_num_of_vertices = d_graph.num_rows;

	DVector d_distance(k_num_of_vertices);
	thrust::fill(thrust::device, d_distance.begin(), d_distance.end(), INT_MAX);

	DVector d_near_set;
	DVector d_far_pile;

	DVector d_out_edges_count(k_num_of_vertices + 1);
	thrust::adjacent_difference(thrust::device, d_graph.row_offsets.begin(),
								d_graph.row_offsets.end(), d_out_edges_count.begin());
	d_out_edges_count.erase(d_out_edges_count.begin());

	DVector d_current_out_edges_count(k_num_of_vertices);
	DVector d_current_out_edges_index(k_num_of_vertices);

	d_distance[0] = 0;
	d_near_set.push_back(0);

	bool is_work_left = true;

	int i = 0;
	while (is_work_left) {
		int upper_distance = (i + 1) * k_delta;
#if DEBUG_MSG
		std::cout << "Iteration #: " << i << "\nUpper Distance: " << upper_distance << std::endl;
#endif
		while (!d_near_set.empty()) {
			// Copy current vertices set count of outgoing edges
			thrust::fill(thrust::device, d_current_out_edges_count.begin(), d_current_out_edges_count.end(), 0);
			thrust::copy(thrust::device,
					thrust::make_permutation_iterator(d_out_edges_count.begin(),
												      d_near_set.begin()),
					thrust::make_permutation_iterator(d_out_edges_count.begin(),
												      d_near_set.end()),
					thrust::make_permutation_iterator(d_current_out_edges_count.begin(),
													  d_near_set.begin()));

			// Get number of outgoing edges
			int num_out_edges = thrust::reduce(thrust::device,
											   d_current_out_edges_count.begin(),
											   d_current_out_edges_count.end());

			// Scan count to get map
			thrust::exclusive_scan(thrust::device,
								   d_current_out_edges_count.begin(),
								   d_current_out_edges_count.end(),
								   d_current_out_edges_index.begin());

			// Expanded source vertices
			DVector d_exp_source_vertices(num_out_edges);
			cusp::offsets_to_indices(d_current_out_edges_index, d_exp_source_vertices);

			// Scatter to put row offset in position
			DVector d_frontier_position(num_out_edges, 1);
			thrust::scatter_if(thrust::device,
							   d_graph.row_offsets.begin(),
					  	  	   d_graph.row_offsets.end() - 1,
					  	  	   d_current_out_edges_index.begin(),
					  	  	   d_current_out_edges_count.begin(),
					  	  	   d_frontier_position.begin());

			// Inclusive scan by key to sum get final frontier positions and values
			thrust::inclusive_scan_by_key(thrust::device,
										  d_exp_source_vertices.begin(),
										  d_exp_source_vertices.end(),
										  d_frontier_position.begin(),
										  d_frontier_position.begin());

			// Tentative distance
			DVector d_tent_distance(num_out_edges);
			thrust::transform(thrust::device,
					thrust::make_permutation_iterator(d_graph.values.begin(),
													  d_frontier_position.begin()),
					thrust::make_permutation_iterator(d_graph.values.begin(),
													  d_frontier_position.end()),
					thrust::make_permutation_iterator(d_distance.begin(),
													  d_exp_source_vertices.begin()),
					d_tent_distance.begin(),
					thrust::plus<int>());

			// Split frontier vertices between near and far pile
			int count_current_near = thrust::count_if(thrust::device,
					d_tent_distance.begin(), d_tent_distance.end(), is_light(upper_distance));
			int count_current_far = d_tent_distance.size() - count_current_near;

			// Resize near and far pile
			d_near_set.clear();
			d_near_set.resize(count_current_near);
			d_far_pile.resize(d_far_pile.size() + count_current_far);

			// Separate indices for near and far pile
			DVector near_frontier_idx(count_current_near);
			DVector d_near_tent_distance(count_current_near);
			DVector far_frontier_idx(count_current_far);
			DVector d_far_tent_distance(count_current_far);
			thrust::partition_copy(thrust::device,
			      make_zip_iterator(make_tuple(d_frontier_position.begin(), d_tent_distance.begin())),
			      make_zip_iterator(make_tuple(d_frontier_position.end(), d_tent_distance.end())),
			      make_zip_iterator(make_tuple(near_frontier_idx.begin(), d_near_tent_distance.begin())),
			      make_zip_iterator(make_tuple(far_frontier_idx.begin(), d_far_tent_distance.begin())),
			      is_light_tuple(upper_distance));

			// Copy vertices to far pile
			thrust::copy(thrust::device,
					thrust::make_permutation_iterator(d_graph.column_indices.begin(), far_frontier_idx.begin()),
					thrust::make_permutation_iterator(d_graph.column_indices.begin(), far_frontier_idx.end()),
					(d_far_pile.end() - count_current_far));

			// Copy vertices to near set
			thrust::copy(thrust::device,
					thrust::make_permutation_iterator(d_graph.column_indices.begin(), near_frontier_idx.begin()),
					thrust::make_permutation_iterator(d_graph.column_indices.begin(), near_frontier_idx.end()),
					d_near_set.begin());

			// Update near set distances
			// Remove if tent_dist[frontier vertex] > dist[frontier vertex]
			ZipTupleTentDistIterator zip_removed_end;
			zip_removed_end = thrust::remove_if(thrust::device,
					thrust::make_zip_iterator(thrust::make_tuple(
							d_near_set.begin(),
							d_near_tent_distance.begin())),
					thrust::make_zip_iterator(thrust::make_tuple(
							d_near_set.end(),
							d_near_tent_distance.end())),
					is_tent_dist_greater(d_distance.data().get()));

			TupleTentDist tent_dist_iter = zip_removed_end.get_iterator_tuple();
			d_near_set.erase(thrust::get<0>(tent_dist_iter), d_near_set.end());
			d_near_tent_distance.erase(thrust::get<1>(tent_dist_iter), d_near_tent_distance.end());

			// Sort for increasing distance
			thrust::sort_by_key(d_near_tent_distance.begin(),
					d_near_tent_distance.end(),
					d_near_set.begin());

			// Get key together maintaining increasing distance within pack of keys
			thrust::stable_sort_by_key(d_near_set.begin(),
					d_near_set.end(),
					d_near_tent_distance.begin());

			// Get unique target vertices (they should have the minimum distance)
			thrust::pair<DVectorIterator, DVectorIterator> new_end;
			new_end = thrust::unique_by_key(d_near_set.begin(),
					d_near_set.end(),
					d_near_tent_distance.begin());

			d_near_set.erase(new_end.first, d_near_set.end());
			d_near_tent_distance.erase(new_end.second, d_near_tent_distance.end());

			// Update distances
			thrust::copy(thrust::device,
					d_near_tent_distance.begin(),
					d_near_tent_distance.end(),
					make_permutation_iterator(d_distance.begin(), d_near_set.begin()));
		}

		is_work_left = !d_far_pile.empty();
		if (is_work_left) {
			// Compact (Remove vertices non-set start distance)
			DVectorIterator new_far_pile_end;
			new_far_pile_end = thrust::remove_if(thrust::device, d_far_pile.begin(),
												 d_far_pile.end(), is_dist_non_set(d_distance.data().get()));
			d_far_pile.erase(new_far_pile_end, d_far_pile.end());

			// Compact (Remove duplicates)
			thrust::sort(thrust::device, d_far_pile.begin(), d_far_pile.end());
			new_far_pile_end = thrust::unique(thrust::device, d_far_pile.begin(), d_far_pile.end());
			d_far_pile.erase(new_far_pile_end, d_far_pile.end());

			// Copy all far to near, it will be split in the inner loop
			d_near_set.resize(d_far_pile.size());
			thrust::copy(thrust::device, d_far_pile.begin(), d_far_pile.end(), d_near_set.begin());
			d_far_pile.clear();

			// Go to next upper distance
			i++;
		}
	}

	std::cout << "Distances: " << std::endl;
	thrust::copy(d_distance.begin(), d_distance.end(), std::ostream_iterator<int>(std::cout, " "));
	std::cout << std::endl;

	return 0;
}


